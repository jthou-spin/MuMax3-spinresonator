#include "hip/hip_runtime.h"

#include <stdint.h>
#include "float3.h"
#include "amul.h"

// Add resonator field to B.
extern "C" __global__ void
addresonatorfield(float* __restrict__  Bx, float* __restrict__  By, float* __restrict__  Bz,
                       float voltage,
                       float current, 
                       float brf,
                       int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {

        Bx[i] += brf * current;
    }
}

